#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int* c) {
	*c = a + b;
}
int main() {
	int a, b, c;
	int* dev_c;
	printf("Nhap he so a: ");
	scanf("%d", &a);
	printf("Nhap he so b: ");
	scanf("%d", &b);
	hipMalloc((void**)&dev_c, sizeof(int));
	add << <1, 128 >> > (a, b, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("The sum of %d and %d is %d\n", a, b, c);
	hipFree(dev_c);
	return 0;
}
