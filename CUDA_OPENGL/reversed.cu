#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void reverseNumber(int n, int* reversed) {
	*reversed = 0;
	while (n != 0)
	{
		*reversed = *reversed * 10 + n % 10;
		n /= 10;
	}
}
int main() {
	int n, reversed;
	int* dev_reversed;
	printf("Nhap he so a: ");
	scanf("%d", &n);
	hipMalloc((void**)&dev_reversed, sizeof(int));
	reverseNumber << <1, 1 >> > (n, dev_reversed);
	hipMemcpy(&n, dev_reversed, sizeof(int), hipMemcpyDeviceToHost);
	printf("So dao nguoc la: %d\n", reversed);
	hipFree(dev_reversed);
	return 0;
}
