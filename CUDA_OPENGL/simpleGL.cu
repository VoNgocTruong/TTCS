#include "hip/hip_runtime.h"
﻿#include <gl/glut.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define PI 3.14159265
struct Vertex {
    float x, y, z;
};
__global__ void polygonKernel(Vertex* vertices, int sides, float radius) {
    for (int i = 0; i < sides; i++) {
        float angle = 2 * PI * i / sides;
        vertices[i].x = radius * cos(angle);
        vertices[i].y = radius * sin(angle);
        vertices[i].z = 0;
    }
}
void display() {
    glClear(GL_COLOR_BUFFER_BIT);
    Vertex* opengl_buffer;
    Vertex* cuda_device_buffer;
    int sides = 10; 
    float radius = 0.5; 
    size_t size = sides * sizeof(Vertex);
    opengl_buffer = (Vertex*)malloc(size);
    hipMalloc((void**)&cuda_device_buffer, size);
    polygonKernel << <1, 1 >> > (cuda_device_buffer, sides, radius);
    hipMemcpy(opengl_buffer, cuda_device_buffer, sides * sizeof(Vertex), hipMemcpyDeviceToHost);
    glBegin(GL_POLYGON);
    for (int i = 0; i < sides; i++) {
        glVertex3f(opengl_buffer[i].x, opengl_buffer[i].y, opengl_buffer[i].z);
    }
    glEnd();
    glFlush();
    hipFree(cuda_device_buffer);
}
int main(int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
    glutInitWindowPosition(80, 80);
    glutInitWindowSize(400, 300);
    glutCreateWindow("A Simple Polygon");
    glutDisplayFunc(display);
    hipSetDevice(0);
    glutMainLoop();
 
}
